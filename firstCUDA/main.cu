#include "hip/hip_runtime.h"
/*
* CURAND API: inizio uso dei PseudoRandom Number Generator
* limitazione a 65535 kernel, lancio monodimensionale/bidimensionale
* attenzione al numero N...non riesco a processare molti dati usando in contemporanea il monitor
*/

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define PI 3.14159265358979323846
#define N 128
#define N2 N

__global__ void setup_kernel ( hiprandStateXORWOW_t * state){
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x+ y*blockDim.x*gridDim.x;
	while (offset<N){
/* Each thread gets same seed , a different sequence number no offset */
		hiprand_init (1234 , offset , 0 , &state[offset]);
		offset += blockDim.x*gridDim.x;
		__syncthreads();
	}
	
}

__global__ void generate_bit_kernel ( hiprandStateXORWOW_t * state , float * result ){
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y*blockDim.x*gridDim.x;
	while (offset < N){
		hiprandStateXORWOW_t localState = state[offset];
		float awgn = hiprand_normal(&localState);
		result[offset] = awgn;
		//state[offset]=localState;
		offset += blockDim.x*gridDim.x;
		__syncthreads();
	}
	
}

int main ( int argc , char * argv []){
	int i;
	//dim3 dimGrid(1024); //numero block
    //dim3 dimBlock(512); //numero threads per block
	dim3 dimGrid(8,8); 
    dim3 dimBlock(16,16); // 2 dim max 512
	hiprandStateXORWOW_t * devStates ;
	float *hostResults, *devResults;
	/* Allocate space for results on host */
	hostResults = ( float *) calloc (N2 , sizeof(float) );
	/* Allocate space for results on device */
	hipMalloc (( void **) &devResults , N2 * sizeof(float) );
	/* Set results to 0 */
	hipMemset ( devResults , 2, N2 * sizeof(float) );
	/* Allocate space for prng states on device */
	hipMalloc (( void **) &devStates , N2 * sizeof(hiprandStateXORWOW_t) );
	/* Setup prng states */
	setup_kernel <<<dimGrid, dimBlock>>>( devStates ) ;
	hipDeviceSynchronize();
	/* Generate and use pseudo - random */
	//for ( i = 0; i < 10; i++) {
		generate_bit_kernel <<<dimGrid,dimBlock>>>( devStates , devResults ) ;
		hipDeviceSynchronize();
	//}
	/* Copy device memory to host */
	hipMemcpy ( hostResults , devResults , N2 * sizeof(float) , hipMemcpyDeviceToHost ) ;
	/* Show result */
	float tmp;
	for ( i=0; i < N; i++ ) {
		tmp+=hostResults[i];
	}
	printf("%f \n",tmp);
	/* Cleanup */
	hipFree(devStates);
	hipFree(devResults);
	free(hostResults);
	system("pause");
	return EXIT_SUCCESS;
}